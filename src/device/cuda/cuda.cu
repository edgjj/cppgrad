#include "cppgrad/device/cuda/cuda.hpp"

// #include <cstring>
// #include <memory>

namespace cppgrad {

std::byte* CUDA::allocate(std::size_t count, std::align_val_t alignment)
{
    std::byte* ptr;
    hipMalloc(&ptr, count);
    return ptr;
}

void CUDA::deallocate(std::byte* ptr, std::align_val_t alignment)
{
    hipFree(ptr);
}

void CUDA::copy(std::byte* from, std::byte* to, std::size_t count)
{
    hipMemcpy(to, from, count, hipMemcpyKind::hipMemcpyDeviceToDevice);
}

void CUDA::copyFromHost(std::byte* from, std::byte* to, std::size_t count)
{
    hipMemcpy(to, from, count, hipMemcpyKind::hipMemcpyHostToDevice);
}

void CUDA::copyToHost(std::byte* from, std::byte* to, std::size_t count)
{
    hipMemcpy(to, from, count, hipMemcpyKind::hipMemcpyDeviceToHost);
}

}