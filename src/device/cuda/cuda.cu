#include "cppgrad/device/cuda/cuda.hpp"
#include "cppgrad/device/cuda/fill_kernel.cuh"

namespace cppgrad {

std::byte* CUDA::allocate(std::size_t count, std::align_val_t alignment, std::string& err)
{
    std::byte* ptr;

    auto result = hipMalloc(&ptr, count);
    if (result != hipSuccess) {
        // retrieve available mem to print out later
        size_t free_mem = 0, total_mem = 0;
        hipMemGetInfo(&free_mem, &total_mem);

        err += "[ ";
        err += type();
        err += " ]";
        err += "Device out of memory. Tried to allocate: ";
        err += std::to_string(count);
        err += " bytes. ";
        err += "Available memory: ";
        err += std::to_string(free_mem);
        err += " bytes.";
        return nullptr;
    }

    return ptr;
}

void CUDA::deallocate(std::byte* ptr, std::align_val_t alignment)
{
    hipDeviceSynchronize();
    hipFree(ptr);
}

void CUDA::copy(std::byte* from, std::byte* to, std::size_t count)
{
    hipMemcpy(to, from, count, hipMemcpyKind::hipMemcpyDeviceToDevice);
}

void CUDA::copy_from_host(std::byte* from, std::byte* to, std::size_t count)
{
    hipMemcpy(to, from, count, hipMemcpyKind::hipMemcpyHostToDevice);
}

void CUDA::copy_to_host(std::byte* from, std::byte* to, std::size_t count)
{
    hipMemcpy(to, from, count, hipMemcpyKind::hipMemcpyDeviceToHost);
}

void CUDA::assign(std::byte* pos, std::byte* value, DType type, std::size_t count)
{
    copy(value, pos, type_size(type) * count);
}

void CUDA::fill(std::byte* pos, std::byte* value, DType type, std::size_t count)
{
    FOREACH_TYPE(type, impl::fill_impl, pos, value, count);
}

std::string_view CUDA::type()
{
    return "cuda"; // include device number there later like cuda:X
}

int CUDA::num_devices()
{
    int nDevices = 0;
    hipGetDeviceCount(&nDevices);

    return nDevices;
}

}