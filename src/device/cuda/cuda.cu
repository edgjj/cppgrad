#include "cppgrad/device/cuda/cuda.hpp"

namespace cppgrad {

std::byte* CUDA::allocate(std::size_t count, std::align_val_t alignment)
{
    std::byte* ptr;
    hipMalloc(&ptr, count);
    return ptr;
}

void CUDA::deallocate(std::byte* ptr, std::align_val_t alignment)
{
    hipFree(ptr);
}

void CUDA::copy(std::byte* from, std::byte* to, std::size_t count)
{
    hipMemcpy(to, from, count, hipMemcpyKind::hipMemcpyDeviceToDevice);
}

void CUDA::copy_from_host(std::byte* from, std::byte* to, std::size_t count)
{
    hipMemcpy(to, from, count, hipMemcpyKind::hipMemcpyHostToDevice);
}

void CUDA::copy_to_host(std::byte* from, std::byte* to, std::size_t count)
{
    hipMemcpy(to, from, count, hipMemcpyKind::hipMemcpyDeviceToHost);
}

int CUDA::num_devices()
{
    int nDevices = 0;
    hipGetDeviceCount(&nDevices);

    return nDevices;
}

}